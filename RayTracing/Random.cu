#include "hip/hip_runtime.h"
#include "Random.h"

__global__ void initRNG(hiprandState *const rngStates, const unsigned int seed) {
  // Determine thread ID
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // Initialise the RNG
  hiprand_init(seed, tid, 0, &rngStates[tid]);
}

extern "C" void init_rng(uint32_t thread_block_count, uint32_t thread_block_size, hiprandState* const rngStates, const unsigned int seed)
{
	initRNG<<<dim3(thread_block_count), dim3(thread_block_size)>>>(rngStates, seed);
}

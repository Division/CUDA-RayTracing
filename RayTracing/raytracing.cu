#include "hip/hip_runtime.h"
#include "RayTracing.h"
#include "Scene.h"
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include "utils/CUDAHelper.h"

using namespace std::chrono;

extern "C" 
{
	void raytracing_process(void* surface, void* surface_last_frame, int width, int height, size_t pitch, int frame_index, RayTracing::Scene* scene);
}


namespace RayTracing
{

	__global__ void initRNG(hiprandState *const rngStates, const unsigned int seed) {
	  // Determine thread ID
	  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	  // Initialise the RNG
	  hiprand_init(seed, tid, 0, &rngStates[tid]);
	}

	CUDARayTracer::CUDARayTracer(SurfaceData surface)
		: surface(surface)
	{
		scene = std::make_unique<Scene>();
		scene->AddSphere(vec3(0, 0, -1), 0.5f);
		scene->AddSphere(vec3(0, -100.5f, -1), 100);
	}

	CUDARayTracer::~CUDARayTracer() = default;
	
	void CUDARayTracer::Process()
	{
		if (!rng_state)
		{
			const auto pixel_count = surface.width * surface.height;
			const dim3 thread_block_size(128);
			const dim3 thread_block_count((pixel_count + thread_block_size.x - 1) / thread_block_size.x);
			rng_state = std::make_unique<CUDAHelper::DeviceMemory>(sizeof(hiprandState) * thread_block_count.x * thread_block_size.x); 
			const uint32_t ms = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
			initRNG<<<thread_block_count, thread_block_size>>>(static_cast<hiprandState*>(rng_state->memory), 0xDEADBEEFu * ms);
		}

		scene->Upload(static_cast<hiprandState*>(rng_state->memory));
		raytracing_process(surface.surface, surface.last_frame_surface, surface.width, surface.height, surface.pitch, frame_index, scene.get());
		CUDA_CHECK(hipMemcpy(surface.last_frame_surface, surface.surface, surface.pitch * surface.height, hipMemcpyDeviceToDevice));
		frame_index++;
	}
}